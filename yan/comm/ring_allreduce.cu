#include "hip/hip_runtime.h"
#include <__clang_cuda_builtin_vars.h>
#include <bits/getopt_core.h>
#include <cassert>
#include <host/nvshmem_api.h>
#include <host/nvshmemx_coll_api.h>
#include <iostream>
#include <nvshmem.h>
#include <nvshmemx.h>

#undef CUDA_CHECK
#define CUDA_CHECK(stmt)                                                          \
    do                                                                            \
    {                                                                             \
        hipError_t result = (stmt);                                              \
        if (hipSuccess != result)                                                \
        {                                                                         \
            fprintf(stderr, "[%s:%d] cuda failed with %s \n", __FILE__, __LINE__, \
                    hipGetErrorString(result));                                  \
            exit(-1);                                                             \
        }                                                                         \
    } while (0)

/* atol() + optional scaled suffix recognition: 1K, 2M, 3G, 1T */
static inline int atol_scaled(const char *str, size_t *out)
{
    int    scale, n;
    double p = -1.0;
    char   f;
    n = sscanf(str, "%lf%c", &p, &f);

    if (n == 2)
    {
        switch (f)
        {
        case 'k':
        case 'K':
            scale = 10;
            break;
        case 'm':
        case 'M':
            scale = 20;
            break;
        case 'g':
        case 'G':
            scale = 30;
            break;
        case 't':
        case 'T':
            scale = 40;
            break;
        default:
            return 1;
        }
    } else if (p < 0)
    {
        return 1;
    } else
        scale = 0;

    *out = (size_t)ceil(p * (1lu << scale));
    return 0;
}

size_t min_size          = 1024 * 1024 * 32;
size_t max_size          = min_size * 16;
size_t num_blocks        = 32;
size_t threads_per_block = 512;
size_t iters             = 4;
size_t warmup_iters      = 1;
size_t step_factor       = 2;
size_t chunk_size        = 262144; // 256K

__global__ void ring_allreduce_kernel(int *dst, const int *src, size_t n_reduce, uint64_t *signal,
                                      size_t chunk_size)
{
    int mype  = nvshmem_my_pe();
    int n_pes = nvshmem_n_pes();
    int peer  = (mype + 1) % n_pes;

    int    thread_idx      = threadIdx.x;
    int    num_threads     = blockDim.x;
    int    num_blocks      = gridDim.x;
    int    block_idx       = blockIdx.x;
    size_t elems_per_block = n_reduce / num_blocks;

    if (elems_per_block * (blockIdx.x + 1) > n_reduce)
    {
        return;
    }
    src    = src + block_idx * elems_per_block;
    dst    = dst + block_idx * elems_per_block;
    signal = signal + block_idx;

    size_t chunk_elems = chunk_size / sizeof(int);
    size_t num_chunks  = elems_per_block / chunk_elems;

    // reduce phase
    for (size_t chunk_idx = 0; chunk_idx < num_chunks; ++chunk_idx)
    {
        if (mype != 0)
        {
            if (thread_idx == 0)
            {
                nvshmem_signal_wait_until(signal, NVSHMEM_CMP_GE, chunk_idx + 1);
            }
            __syncthreads();
            for (size_t i = thread_idx; i < chunk_elems; i += num_threads)
            {
                dst[i] = dst[i] + src[i];
            }
            __syncthreads();
        }
        if (thread_idx == 0)
        {
            nvshmem_int_put_signal_nbi(dst, (mype == 0) ? src : dst, chunk_elems,
                                       signal, 1, NVSHMEM_SIGNAL_ADD, peer);

            src = src + chunk_elems;
            dst = dst + chunk_elems;
        }

        dst = dst - num_chunks * chunk_elems;
        if (thread_idx == 0)
        {
            for (size_t chunk_idx = 0; chunk_idx < num_chunks; ++chunk_idx)
            {
                if (mype < n_pes - 1)
                {
                    nvshmem_signal_wait_until(signal, NVSHMEM_CMP_GE,
                                              (mype == 0) ? chunk_idx + 1 : num_chunks + chunk_idx + 1);
                }
                if (mype < n_pes - 2)
                {
                    nvshmem_int_put_signal_nbi(dst, dst, chunk_elems, signal, 1, NVSHMEM_SIGNAL_ADD, peer);
                }
                dst = dst + chunk_elems;
            }
            *signal = 0;
        }
    }
}

int main(int argc, char *argv[])
{
    int c;
    while ((c = getopt(argc, argv, "b:e:f:n:w:c:t:m:")) != -1)
    {
        switch (c)
        {
        case 'b':
            atol_scaled(optarg, &min_size);
            break;
        case 'e':
            atol_scaled(optarg, &max_size);
            break;
        case 'f':
            atol_scaled(optarg, &step_factor);
            break;
        case 'n':
            atol_scaled(optarg, &iters);
            break;
        case 'w':
            atol_scaled(optarg, &warmup_iters);
            break;
        case 'c':
            atol_scaled(optarg, &num_blocks);
            break;
        case 't':
            atol_scaled(optarg, &threads_per_block);
            break;
        case 'm':
            atol_scaled(optarg, &chunk_size);
            break;
        case '?':
            if (optopt == 'c')
                fprintf(stderr, "Option -%c requires an argument.\n", optopt);
            else if (isprint(optopt))
                fprintf(stderr, "Unknown option `-%c'.\n", optopt);
            else
                fprintf(stderr, "Unknown option character `\\x%x'.\n", optopt);
            return 1;
        default:
            abort();
        }
    }
    size_t min_ints = min_size / sizeof(int);
    assert(min_ints % num_blocks == 0);

    nvshmem_init();

    int          mype      = nvshmem_my_pe();
    int          n_pes     = nvshmem_n_pes();
    int          mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipStream_t stream;
    hipEvent_t  start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipSetDevice(mype_node));
    CUDA_CHECK(hipStreamCreate(&stream));

    size_t    max_ints = max_size / sizeof(int);
    int      *dst      = (int *)nvshmem_malloc(max_size);
    int      *src      = (int *)nvshmem_malloc(max_size);
    int      *data_h   = (int *)malloc(max_size);
    uint64_t *signal   = (uint64_t *)nvshmem_calloc(num_blocks, sizeof(uint64_t));

    dim3 gridDim(num_blocks);
    dim3 blockDim(threads_per_block);

    for (size_t i = 0; i < max_ints; ++i)
    {
        data_h[i] = i;
    }

    CUDA_CHECK(hipMemcpyAsync(src, data_h, max_size, hipMemcpyHostToDevice, stream));
    nvshmemx_barrier_all_on_stream(stream);

    for (size_t size = min_size; size <= max_size; size *= step_factor)
    {
        size_t num_ints = size / sizeof(int);
        void  *args[]   = {&dst, &src, &num_ints, &signal, &chunk_size};

        // do warmup
        for (size_t i = 0; i < warmup_iters; ++i)
        {
            nvshmemx_collective_launch((const void *)ring_allreduce_kernel, gridDim, blockDim, args, 0, stream);
            nvshmemx_barrier_all_on_stream(stream);
        }
        CUDA_CHECK(hipStreamSynchronize(stream));

        CUDA_CHECK(hipEventRecord(start, stream));
        for (size_t i = 0; i < iters; ++i)
        {
            nvshmemx_collective_launch((const void *)ring_allreduce_kernel, gridDim, blockDim, args, 0, stream);
            nvshmemx_barrier_all_on_stream(stream);
        }
        CUDA_CHECK(hipEventRecord(stop, stream));

        CUDA_CHECK(hipStreamSynchronize(stream));
        if (mype == 0)
        {
            float ms;
            CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
            std::cout << "Size: " << size << " bytes, "
                      << "Time: " << ms / iters << " ms, "
                      << "Bandwidth: " << (size * iters) / (ms / 1000.0) / (1024 * 1024) << " MB/s" << std::endl;
        }

        CUDA_CHECK(hipMemcpy(data_h, dst, size, hipMemcpyDeviceToHost));
        for (size_t i = 0; i < num_ints; ++i)
        {
            if (data_h[i] != (int)i * n_pes)
            {
                std::cerr << "Error at index " << i << ": expected " << (int)i * n_pes
                          << ", got " << data_h[i] << std::endl;
                exit(-1);
            }
        }
    }

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    nvshmem_free(dst);
    nvshmem_free(src);
    nvshmem_free(signal);
    free(data_h);

    nvshmem_finalize();
    return 0;
}